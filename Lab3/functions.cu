#include "hip/hip_runtime.h"
#include "functions.h"

/*
Descripción: Función del host que abre la imagen y la almacena en un arreglo
Entrada: Nombre del archivo que contiene la imagen, buffer en donde los valores serán 
         almacenados, ancho y largo de la imagen.
Proceso: con fopen se abre la imagen, se lee y se almacena en buffer_out.
Salida: Arreglo con los valores de la imagen.
*/
__host__ void read_image(char* file_name, unsigned short int * buffer_out, int M, int N){
    FILE* image_raw = fopen(file_name, "rb");
    fread(buffer_out, sizeof(unsigned short int), M*N, image_raw);
    
}

/*
Descripción: La función calcula el histograma de la imagen en memoria global.
Entrada: Buffer con los valores de la imagen, histograma en donde almacenarán los datos
         y dimensiones de la imagen. 
Proceso: Se calcula el id global de la hebra y mediante atomicAdd se suma 1 en cada posición
         del histograma cuando la imagen tenga ese valor. Se utiliza atomic debido a que asegura
         que no exista problema al acceder a memoria. 
         Un aspecto que se considera es que el valor que toma la hebra global no puede ser mayor al tamaño
         de la imagen.
Salida: Histograma con las frecuencias de los valores de la imagen
*/
__global__ void histgmem(unsigned short int* buffer, int* histogram, int image_length){
    
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned short int buffer_id = buffer[id];
    if(id<image_length){
        atomicAdd(&histogram[buffer_id],1);
    }
}

/*
Descripción: La función calcula el histograma de la imagen en memoria compartida.
Entrada: Buffer con los valores de la imagen, histograma en donde almacenarán los datos
         y dimensiones de la imagen.
Proceso: Se calcula el id global de la hebra y local. Además se crea un arreglo de histogramas temporal
         para cada uno de los bloques. Luego, mediante atomicAdd se suma 1 en cada posición
         del histograma cuando la imagen tenga ese valor. A continuación se usa syncthreads para que todos
         los bloques terminen su ejecución y finalmente se utiliza otra vez atomiAdd, pero esta vez para 
         sumar todos los valores almacenados en los histogramas compartidos en un histograma global. 
         Un aspecto que se considera es que el valor que toma la hebra global no puede ser mayor al tamaño
         de la imagen.
Salida: Histograma con las frecuencias de los valores de la imagen
*/
__global__ void histsmem(unsigned short int* buffer, int* histogram, int image_length){
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int j = threadIdx.x;
    __shared__ int temporal[256];
    if(j == 0){
        for(int l = 0; l<256; l++)
            temporal[l] = 0;
    }
    __syncthreads();
    if(id<image_length){
        atomicAdd(&temporal[buffer[id]],1);
    }
    __syncthreads();
    if(j == 0){
        for(int l = 0; l<256; l++)
            atomicAdd(&histogram[l],temporal[l]);
    }
    

}
/*
Descripción: Se muestra por pantalla la frecuencia de cada pixel para el histograma con memoria global y compartida.
Entrada: Histograma obtenido con memoria global y compartida.
Proceso: Se recorre cada elemento de los histogramas y se muestra por pantalla.
Salida: Print de los histogramas obtenidos con memoria global y compartida
*/
__host__ void debug( int * hist_final, int * hist_final2){
        for(int i= 0; i < 256; i++){
            printf("%d  %d\n", hist_final[i], hist_final2[i]);
        }
    }


/*
Descripción: Función del host escribe un archivo de texto con los resultados de los histogramas generados
Entrada: Nombre del archivo a escribir, y los histogramas a escribir en el archivo 
Proceso: Se crea un archivo de salida con el nombre de file_name que se ingresa en la entrada
         y se almacenan los valores de los histogramas en dicho archivo.
salida: Archivo de texto con los histogramas.
*/
__hos
__host__ void write_histogram(char* file_name, int * histogram1, int * histogram2){
    FILE* out_file = fopen(file_name, "wb");\
    for(int i = 0; i<256; i++){
        fprintf(out_file, "%d %d\n", histogram1[i], histogram2[i]);
    }
    fclose(out_file);
}
